#include "triplet.h"
#include <stdio.h>
#include <stdlib.h>
#include "cutil_inline.h"
#include "circuit_host.h"
#include "circuit_kernel.h"
#include "block.h"
#include <iostream>
#include <fstream>
using namespace std;

const unsigned int WARPSIZE = 32;
texture <float, 1, hipReadModeElementType> L_tex;
hipChannelFormatDesc channelDesc = 
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	
void substitute_CK_host(cholmod_factor *L, cholmod_dense *b, cholmod_dense *&x){
	hipArray *L_d = NULL;
	double *b_x_d = NULL;
	// copy data from L to hipArray L_d
	// bind L_d into texture memory
	substitute_setup(L, L_d, b, x, b_x_d);
	
	dim3 dimGrid(1, 1);
	dim3 dimBlock(WARPSIZE, 1, 1);
	// perform for- and back-ward substitution for each block
	// solution will be written from shared memory into global memory
	substitute_CK_kernel<<<dimGrid, dimBlock>>>(L_d, b_x_d);

	// copy solution back from GPU into CPU
	// where CPU will perform the find_diff and updaterhs()
	substitute_copy_back(x, b_x_d, b->nzmax);
	substitute_CK_free(L_d, b_x_d);
}

// copy data from host to device side
// 1. load sparse matrix L from host into global memory, including 4
//    sub-arrays: row index, column index, nz in each column, 
//    and value
// 2. combine L into texture memory
// 3. load dense array b and x from host into global 1d array
void substitute_setup(cholmod_factor *L, hipArray *L_d, cholmod_dense *b, cholmod_dense *&x){
	// count is the total bytes of all needed data in L
	// including in order of nz, col, row, and x
	size_t count = sizeof(size_t)*(L->n + (L->n+1)+ L->nzmax);
	count +=sizeof(double)*L->nzmax;
	
	// allocate hipArray and bind it with texture
	hipMallocArray(&L_d, &L_tex.channelDesc, count, 1);

	size_t index = 0;
	hipMemcpyToArray(L_d, index, 0, L->nz, sizeof(size_t)*L->n, 
		hipMemcpyHostToDevice);
	index += L->n;
	hipMemcpyToArray(L_d, index, 0, L->p, sizeof(size_t)*(L->n+1), 
		hipMemcpyHostToDevice);
	index += L->n + 1;
	hipMemcpyToArray(L_d, index, 0, L->i, sizeof(size_t)*L->nzmax, 
		hipMemcpyHostToDevice);
	index += L->nzmax;
	hipMemcpyToArray(L_d, index, 0, L->x, sizeof(double)*L->nzmax, 
		hipMemcpyHostToDevice);

	// bind L_d to texture memory
	hipBindTextureToArray(L_tex, L_d, channelDesc);	
	
	// malloc b and x into a 1d array, copy from host to device
	count = sizeof(double)* 2 * b->nzmax;
	hipMalloc((void**)&b_x_d, count);
	index = 0;
	hipMemcpy(&b_x_d[0], b->x, b->nzmax, hipMemcpyHostToDevice);
	index += b->nzmax;
	hipMemcpy(&b_x_d[index], x->x, x->nzmax, hipMemcpyHostToDevice);
}

void substitute_copy_back(cholmod_dense *x_h, double *b_x_d, size_t index){
	size_t count = index; // both are nzmax
	hipMemcpy(&x_h->x[0], b_x_d, count, hipMemcpyDeviceToHost);
}

void substitute_CK_free(hipArray *L_d, double *b_x_d){
	hipFree(b_x_d);
	hipFree(L_d);
}
