#include "hip/hip_runtime.h"
#include "triplet.h"
#include "trip_L.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "circuit_host.h"
#include <circuit_kernel.h>
#include "block.h"
#include "global.h"
#include <iostream>
#include <fstream>
using namespace std;

texture <float, 1, hipReadModeElementType> L_tex;
hipChannelFormatDesc channelDesc = 
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

// kernel function, doing forward and backward substitution
// data stored in L_d: row, col, and val
// data stored in b_x_d: b and x 
/*__global__ void substitute_CK_kernel(float *L_d, size_t L_h_nz, float *b_x_d, size_t n){
	int tid = threadIdx.x;
	// load data into shared memory
	extern __shared__ float b_x_s[];

	int i, j;
	int iter = n / blockDim.x ;
	if((n % blockDim.x)!=0) iter += 1;
	for(i=0; i< iter; i++){
		int base = i * blockDim.x;
		if((base+tid) < n)
			b_x_s[base+tid] = b_x_d[base+tid];
	}
	__syncthreads();
		
	i = 0; j = 0;
	int index_col = 0, index_row = 0;
	size_t row_p;
	// tid < WARPSIZE will do substitution
	// then all threads will copy the solution 
	// from shared memory into global memory
	if(tid < HALF_WARP){
		// doing forward substitution
		while(i < 3*L_h_nz){
			row_p = tex1Dfetch(L_tex, i);
		
			// xj = bj / Ajj
			index_row = tex1Dfetch(L_tex, i);
			b_x_s[index_row] /= tex1Dfetch(L_tex, i+2);
			
			j = i+3;
			if(j >= 3 * L_h_nz) break;
			//while(L_d[j] != L_d[j+1]){
			while(tex1Dfetch(L_tex, j) != tex1Dfetch(L_tex, j+1)){
				// bi = bi - Aij * xj
				index_row = tex1Dfetch(L_tex, j);
				index_col = tex1Dfetch(L_tex, j+1);
				b_x_s[index_row] -= tex1Dfetch(L_tex, j+2) * b_x_s[index_col];
				j += 3;
			}
			i = j;
		}
		
		// doing backward substitution
		i = 3 * L_h_nz - 3;
		while(i >= 0){
			row_p = tex1Dfetch(L_tex, i);
				
			// xi = bi / Aij
			b_x_s[row_p] /= tex1Dfetch(L_tex, i+2);

			j = i-3;
			if(j<0) break;
			//while(L_d[j]!=L_d[j+1]){
			while(tex1Dfetch(L_tex, j) != tex1Dfetch(L_tex, j+1)){
				// bi = bi - Aij * xj
				index_row = tex1Dfetch(L_tex, j);
				index_col = tex1Dfetch(L_tex, j+1);
				b_x_s[index_col] -= tex1Dfetch(L_tex, j+2) * b_x_s[index_row];
				j -= 3;
			}
			i = j;
		}
	}
		
	// after computing, copy back into global memory
	for(i=0; i< iter; i++){
		int base = i * blockDim.x;
		if((base+tid) < n)
			//b_x_d[base+tid] = tex1Dfetch(L_tex, tid);
			b_x_d[base+tid] = b_x_s[base+tid];
	}
}*/

void substitute_CK_host(float *L_h, size_t L_h_nz, float *bp, float *xp, size_t n){
	float *L_d ;	
	float *b_x_d ;
	// dump hipMalloc, as the first call wll cost about 1s
	// which is much larger than usual 1e-6s
	hipMalloc((void**)&L_d, sizeof(float));
	unsigned int timer;
	float cudaTime;
	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));

	// copy data from L to hipArray L_d
	// bind L_d into texture memory	
	substitute_setup(L_h, L_h_nz, L_d, bp, xp, b_x_d, n);

	dim3 dimGrid(1, 1);
	dim3 dimBlock(256, 1, 1);
	int sharedMemSize =sizeof(float) *n;
	clog<<"shared mem size: "<<sharedMemSize<<endl;
	// perform for- and back-ward substitution for each block
	// solution will be written from shared memory into global memory
	unsigned int timer_compute;
	float cudaTime_compute;
	CUT_SAFE_CALL(cutCreateTimer(&timer_compute));
	CUT_SAFE_CALL(cutStartTimer(timer_compute));
	
	//substitute_CK_kernel<<<dimGrid, dimBlock, sharedMemSize>>>(L_d, L_h_nz, b_x_d, n);
	
	cutilCheckMsg("Kernel execution failed.");
	CUT_SAFE_CALL(cutStopTimer(timer_compute));
	cudaTime_compute = cutGetTimerValue(timer_compute);
	clog<<"kernel time: "<<cudaTime_compute/1000<<" (s) "<<endl;
	CUT_SAFE_CALL(cutDeleteTimer(timer_compute));	

	// copy solution back from GPU into CPU
	// where CPU will perform the find_diff and updaterhs()
	//cutilSafeCall(hipMemcpy(bp, b_x_d, sizeof(float)*n, hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(xp, b_x_d, sizeof(float)*n, hipMemcpyDeviceToHost));
	
	cutilSafeCall(hipUnbindTexture(L_tex));
	cutilSafeCall(hipFree(L_d));
	cutilSafeCall(hipFree(b_x_d));
	
	CUT_SAFE_CALL(cutStopTimer(timer));
	cudaTime = cutGetTimerValue(timer);
	clog<<"gpu time: "<<cudaTime/1000<<" (s) "<<endl;
	CUT_SAFE_CALL(cutDeleteTimer(timer));
}


// copy data from host to device side
// 1. load sparse matrix L from host into global memory, including 4
//    sub-arrays: row index, column index, nz in each column, 
//    and value
// 2. combine L into texture memory
// 3. load dense array b and x from host into global 1d array
void substitute_setup(float *L_h, size_t L_h_nz, float *&L_d, float *bp, float *xp, float *&b_x_d, size_t n){
	size_t count = sizeof(float) * 3 * L_h_nz;		
	// allocate hipArray and bind it with texture
	hipMalloc((void**)&L_d, count);		
	clog<<"Texture memory size: "<<count<<endl;
	cutilSafeCall(hipMemcpy(L_d, L_h, count, hipMemcpyHostToDevice));	
	cutilSafeCall(hipBindTexture(0, L_tex, L_d, channelDesc, count));
		
	// malloc b and x into a 1d array, copy from host to device
	count = sizeof(float) * n;
	cutilSafeCall(hipMalloc((void**)&b_x_d, count));	
	cutilSafeCall(hipMemcpy(b_x_d, bp, count, hipMemcpyHostToDevice));
}

// block version of substitution
void block_cuda_setup(BlockInfo &block_info, float *&L_d, 
	float *&b_x_d,size_t &total_n, size_t &total_nz,
	int *&L_nz_d, size_t *&base_nz_d, int *&L_n_d, 
	size_t *&base_n_d, size_t &sharedMemSize){

	// dump hipMalloc, as the first call wll cost about 1s
	// which is much larger than usual 1e-6s
	hipMalloc((void**)&L_d, sizeof(float));
	
	/*unsigned int timer;
	float cudaTime;
	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));*/
	
	// copy data from L to hipArray L_d
	// bind L_d into texture memory
	// copy L_n_nz_h into constant memory in 1D array
	substitute_block_setup(block_info, L_d, b_x_d, L_nz_d, 
		L_n_d, base_nz_d, base_n_d, total_n, total_nz);
	
	// find max block size in block_info
	int max_block_size = 0;
	for(size_t i=0;i<block_info.size();i++)
		if(block_info[i].count > max_block_size)
			max_block_size = block_info[i].count;
	sharedMemSize =sizeof(float) *max_block_size;
	clog<<"shared mem size: "<<sharedMemSize<<endl;
}

void block_cuda_iteration(BlockInfo &block_info, float *&L_d, 
	float *&b_x_d, size_t &total_n, size_t &total_nz,
	int *&L_nz_d, size_t *&base_nz_d, int *&L_n_d, 
	size_t *&base_n_d, size_t &sharedMemSize){
	
	// configurate kernel info
	clog<<block_info.X_BLOCKS<<" "<<block_info.Y_BLOCKS<<endl;
	dim3 dimGrid(block_info.X_BLOCKS, block_info.Y_BLOCKS);
	dim3 dimBlock(256, 1, 1);
	
	// copy b_x_d
	size_t base = 0;
	for(size_t i=0;i<block_info.size();i++){
		//clog<<"block id: "<<block_info[i].bid<<endl;
		//for(size_t j=0;j<block_info[i].count;j++)
			//clog<<j<<" "<<block_info[i].bnewp_f[j]<<endl;
		cutilSafeCall(hipMemcpy(&b_x_d[base], block_info[i].bnewp_f, 
		sizeof(float)*block_info[i].count, hipMemcpyHostToDevice));
		base += block_info[i].count;
	}

	/*unsigned int timer_compute;
	float cudaTime_compute;
	CUT_SAFE_CALL(cutCreateTimer(&timer_compute));
	CUT_SAFE_CALL(cutStartTimer(timer_compute));*/
	
	CK_block_kernel<<<dimGrid, dimBlock, sharedMemSize>>>
		(L_d, b_x_d, L_nz_d, L_n_d, base_nz_d, base_n_d);
	
	cutilCheckMsg("Kernel execution failed.");
	/*CUT_SAFE_CALL(cutStopTimer(timer_compute));
	cudaTime_compute = cutGetTimerValue(timer_compute);
	clog<<"kernel time: "<<cudaTime_compute/1000<<" (s) "<<endl;
	CUT_SAFE_CALL(cutDeleteTimer(timer_compute));*/

	// copy solution back from GPU into CPU
	// where CPU will perform the find_diff and updaterhs()
	base = 0;
	for(size_t i=0;i<block_info.size();i++){
		cout<<"block index: "<<i<<endl;
		hipMemcpy(block_info[i].xp_f, 
		&b_x_d[base], sizeof(float)*block_info[i].count, 
		hipMemcpyDeviceToHost);
		base  += block_info[i].count;
		for(size_t j=0;j<block_info[i].count;j++)
			cout<<j<<" "<<block_info[i].xp_f[j]<<endl;
	}	
	
	/*CUT_SAFE_CALL(cutStopTimer(timer));
	cudaTime = cutGetTimerValue(timer);
	clog<<"gpu time: "<<cudaTime/1000<<" (s) "<<endl;
	CUT_SAFE_CALL(cutDeleteTimer(timer));*/
}

void block_cuda_free(float *&L_d, float *&b_x_d, int *&L_nz_d, 
	size_t *&base_nz_d, int *&L_n_d, size_t *&base_n_d){
	cutilSafeCall(hipUnbindTexture(L_tex));
	hipFree(L_d); hipFree(b_x_d);
	hipFree(L_nz_d); hipFree(base_nz_d);
	hipFree(L_n_d); hipFree(base_n_d);
}

// copy data from host to device side
// 1. load sparse matrix L from host into global memory, 
//    including 3 sub-arrays: row index, column index, and value
// 2. combine L into texture memory
// 3. load dense array b and x from host into global 1d array
// 4. copy L_n_nz indo constant memory
void substitute_block_setup(BlockInfo &block_info, float *&L_d, 
	float *&b_x_d, int *&L_nz_d, int *&L_n_d, 
	size_t *&base_nz_d, size_t *&base_n_d, size_t &total_n, 
	size_t &total_nz){
	// L_nz_h stores nz, L_n_h stores n for each block
	int *L_nz_h; size_t *base_nz_h;
	int *L_n_h; size_t *base_n_h;

	L_nz_h = new int [block_info.size()];
	L_n_h = new int [block_info.size()];
	base_nz_h = new size_t [block_info.size()];
	base_n_h = new size_t [block_info.size()];

	for(size_t i=0;i<block_info.size();i++){
		L_n_h[i] = block_info[i].count;
		L_nz_h[i] = block_info[i].L_h_nz;
		if(i==0){
			base_nz_h[i] = 0;
			base_n_h[i] = 0;
		}
		else{
			base_nz_h[i] = base_nz_h[i-1] +
				       3*block_info[i-1].L_h_nz;
			base_n_h[i] = base_n_h[i-1]+
				       block_info[i-1].count;
		} 
	}
	total_n = base_n_h[block_info.size()-1] + 
		  block_info[block_info.size()-1].count;
	total_nz = base_nz_h[block_info.size()-1] + 
		  block_info[block_info.size()-1].L_h_nz;

	// Allocate and copy L_nz_d, as well as L_n_d
	size_t count = sizeof(int)*block_info.size();
	cutilSafeCall(hipMalloc((void**)&L_nz_d, count));
	cutilSafeCall(hipMalloc((void**)&L_n_d, count));
	cutilSafeCall(hipMemcpy(L_nz_d, L_nz_h, count, 
			hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(L_n_d, L_n_h, count, 
			hipMemcpyHostToDevice));
	
	// Allocate and copy base_nz_d, as well as base_n_d
	count = sizeof(size_t) *block_info.size();
	cutilSafeCall(hipMalloc((void**)&base_nz_d, count));
	cutilSafeCall(hipMalloc((void**)&base_n_d, count));
	cutilSafeCall(hipMemcpy(base_nz_d, base_nz_h, count, 
			hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(base_n_d, base_n_h, count, 
			hipMemcpyHostToDevice));
	
	// free host memory
	free(L_nz_h); free(L_n_h);
	free(base_nz_h); free(base_n_h);
		
	// allocate hipArray and bind it with texture
	count = sizeof(float) * 3 * total_nz;
	cutilSafeCall(hipMalloc((void**)&L_d, count));		
	clog<<"Texture memory size: "<<count<<endl;
	size_t base = 0;
	for(size_t i=0;i<block_info.size();i++){
		cutilSafeCall(hipMemcpy(&L_d[base], block_info[i].L_h, 
		sizeof(float)*3*block_info[i].L_h_nz, hipMemcpyHostToDevice));
		base += 3*block_info[i].L_h_nz;
	}
	cutilSafeCall(hipBindTexture(0, L_tex, L_d, channelDesc, count));
		
	// malloc b and x into a 1d array, copy from host to device
	count = sizeof(float) * total_n;
	cutilSafeCall(hipMalloc((void**)&b_x_d, count));
}
