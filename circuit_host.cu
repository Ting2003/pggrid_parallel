#include "hip/hip_runtime.h"
#include "triplet.h"
#include "trip_L.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "circuit_host.h"
//#include <circuit_kernel.h>
#include "block.h"
#include "global.h"
#include <iostream>
#include <fstream>
using namespace std;

texture <float, 1, hipReadModeElementType> L_tex;
hipChannelFormatDesc channelDesc = 
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

// kernel function, doing forward and backward substitution
// data stored in L_d: nz, col, row, and x
// data stored in b_x_d: b and x 
__global__ void substitute_CK_kernel(float *L_d, size_t L_h_nz, float *b_x_d, size_t n){
	int tid = threadIdx.x;
	// load data into shared memory
	extern __shared__ float b_x_s[];

	int i, j;
	int iter = (n << 1) / blockDim.x ;
	if(((n << 1) % blockDim.x)!=0) iter += 1;
	for(i=0; i< iter; i++){
		int base = i * blockDim.x;
		if((base+tid) < 2 * n)
			b_x_s[base+tid] = b_x_d[base+tid];
	}
	__syncthreads();
		
	i = 0; j = 0;
	int index_col = 0, index_row = 0;
	size_t row_p;
	// tid < WARPSIZE will do substitution
	// then all threads will copy the solution 
	// from shared memory into global memory
	if(tid < HALF_WARP){
		// doing forward substitution
		while(i < 3*L_h_nz){
			row_p = tex1Dfetch(L_tex, i);
		
			// xj = bj / Ajj
			index_row = tex1Dfetch(L_tex, i);
			b_x_s[index_row] /= tex1Dfetch(L_tex, i+2);
			
			j = i+3;
			if(j >= 3 * L_h_nz) break;
			//while(L_d[j] != L_d[j+1]){
			while(tex1Dfetch(L_tex, j) != tex1Dfetch(L_tex, j+1)){
				// bi = bi - Aij * xj
				index_row = tex1Dfetch(L_tex, j);
				index_col = tex1Dfetch(L_tex, j+1);
				b_x_s[index_row] -= tex1Dfetch(L_tex, j+2) * b_x_s[index_col];
				j += 3;
			}
			i = j;
		}
		
		// doing backward substitution
		i = 3 * L_h_nz - 3;
		while(i >= 0){
			row_p = tex1Dfetch(L_tex, i);
				
			// xi = bi / Aij
			b_x_s[n+row_p] = b_x_s[row_p] / tex1Dfetch(L_tex, i+2);

			j = i-3;
			if(j<0) break;
			//while(L_d[j]!=L_d[j+1]){
			while(tex1Dfetch(L_tex, j) != tex1Dfetch(L_tex, j+1)){
				// bi = bi - Aij * xj
				index_row = tex1Dfetch(L_tex, j);
				index_col = tex1Dfetch(L_tex, j+1);
				b_x_s[index_col] -= tex1Dfetch(L_tex, j+2) * b_x_s[n+index_row];
				j -= 3;
			}
			i = j;
		}
	}
		
	// after computing, copy back into global memory
	for(i=0; i< iter; i++){
		int base = i * blockDim.x;
		if((base+tid) < 2 * n)
			//b_x_d[base+tid] = tex1Dfetch(L_tex, tid);
			b_x_d[base+tid] = b_x_s[base+tid];
	}
}

void substitute_CK_host(float *L_h, size_t L_h_nz, float *bp, float *xp, size_t n){
	float *L_d ;	
	float *b_x_d ;
	// dump hipMalloc, as the first call wll cost about 1s
	// which is much larger than usual 1e-6s
	hipMalloc((void**)&L_d, sizeof(float));
	unsigned int timer;
	float cudaTime;
	CUT_SAFE_CALL(cutCreateTimer(&timer));
	CUT_SAFE_CALL(cutStartTimer(timer));

	// copy data from L to hipArray L_d
	// bind L_d into texture memory	
	substitute_setup(L_h, L_h_nz, L_d, bp, xp, b_x_d, n);

	dim3 dimGrid(1, 1);
	dim3 dimBlock(256, 1, 1);
	int sharedMemSize =sizeof(float) *2 *n;
	clog<<"shared mem size: "<<sharedMemSize<<endl;
	// perform for- and back-ward substitution for each block
	// solution will be written from shared memory into global memory
	unsigned int timer_compute;
	float cudaTime_compute;
	CUT_SAFE_CALL(cutCreateTimer(&timer_compute));
	CUT_SAFE_CALL(cutStartTimer(timer_compute));
	substitute_CK_kernel<<<dimGrid, dimBlock, sharedMemSize>>>(L_d, L_h_nz, b_x_d, n);
	cutilCheckMsg("Kernel execution failed.");
	CUT_SAFE_CALL(cutStopTimer(timer_compute));
	cudaTime_compute = cutGetTimerValue(timer_compute);
	clog<<"kernel time: "<<cudaTime_compute/1000<<" (s) "<<endl;
	CUT_SAFE_CALL(cutDeleteTimer(timer_compute));	

	// copy solution back from GPU into CPU
	// where CPU will perform the find_diff and updaterhs()
	//substitute_copy_back(x, b_x_d, b->nrow);
	//substitute_CK_free(L_d, b_x_d);
	//cutilSafeCall(hipMemcpy(bp, b_x_d, sizeof(float)*n, hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(xp, &b_x_d[n], sizeof(float)*n, hipMemcpyDeviceToHost));
	cutilSafeCall(hipUnbindTexture(L_tex));
	cutilSafeCall(hipFree(L_d));
	cutilSafeCall(hipFree(b_x_d));
	
	CUT_SAFE_CALL(cutStopTimer(timer));
	cudaTime = cutGetTimerValue(timer);
	clog<<"gpu time: "<<cudaTime/1000<<" (s) "<<endl;
	CUT_SAFE_CALL(cutDeleteTimer(timer));
}


// copy data from host to device side
// 1. load sparse matrix L from host into global memory, including 4
//    sub-arrays: row index, column index, nz in each column, 
//    and value
// 2. combine L into texture memory
// 3. load dense array b and x from host into global 1d array
void substitute_setup(float *L_h, size_t L_h_nz, float *&L_d, float *bp, float *xp, float *&b_x_d, size_t n){
	size_t count = sizeof(float) * 3 * L_h_nz;		
	// allocate hipArray and bind it with texture
	hipMalloc((void**)&L_d, count);		
	cutilSafeCall(hipMemcpy(L_d, L_h, count, hipMemcpyHostToDevice));	
	cutilSafeCall(hipBindTexture(0, L_tex, L_d, channelDesc, count));
		
	// malloc b and x into a 1d array, copy from host to device
	count = sizeof(float)* 2 * n;
	cutilSafeCall(hipMalloc((void**)&b_x_d, count));	
	size_t index = 0;
	cutilSafeCall(hipMemcpy(b_x_d, bp, sizeof(float)*n, hipMemcpyHostToDevice));
	index += n;
	cutilSafeCall(hipMemcpy(&b_x_d[index], xp, sizeof(float)*n, hipMemcpyHostToDevice));
}

/*
void substitute_copy_back(cholmod_dense *x_h, double *b_x_d, size_t index){
	size_t count = index; // both are nzmax
	hipMemcpy(&x_h->x, &b_x_d[count], count, hipMemcpyDeviceToHost);
}

void substitute_CK_free(float *L_d, double *b_x_d){
	hipUnbindTexture(L_tex);
	hipFree(b_x_d);
	hipFree(L_d);
}*/
