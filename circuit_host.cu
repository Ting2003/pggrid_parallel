#include "hip/hip_runtime.h"
#include "triplet.h"
#include "trip_L.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "circuit_host.h"
//#include <circuit_kernel.h>
#include "block.h"
#include "global.h"
#include <iostream>
#include <fstream>
using namespace std;

texture <float, 1, hipReadModeElementType> L_tex;
hipChannelFormatDesc channelDesc = 
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

// kernel function, doing forward and backward substitution
// data stored in L_d: nz, col, row, and x
// data stored in b_x_d: b and x 
__global__ void substitute_CK_kernel(float *L_d, size_t L_h_nz, float *b_x_d, size_t n){
	int tid = threadIdx.x;
	// load data into shared memory
	extern __shared__ float b_x_s[];

	int i, j;
	int iter = (n << 1) / blockDim.x ;
	if(((n << 1) % blockDim.x)!=0) iter += 1;
	for(i=0; i< iter; i++){
		int base = i * blockDim.x;
		if((base+tid) < 2 * n)
			b_x_s[base+tid] = b_x_d[base+tid];
	}
	__syncthreads();
		
	i = 0; j = 0;
	int index_col = 0, index_row = 0;
	size_t row_p;
	// tid < WARPSIZE will do substitution
	// then all threads will copy the solution 
	// from shared memory into global memory
	if(tid < HALF_WARP){
		// doing forward substitution
		while(i < 3*L_h_nz){
			row_p = tex1Dfetch(L_tex, i);
		
			// xj = bj / Ajj
			index_row = tex1Dfetch(L_tex, i);
			b_x_s[index_row] /= tex1Dfetch(L_tex, i+2);
			
			j = i+3;
			if(j >= 3 * L_h_nz) break;
			//while(L_d[j] != L_d[j+1]){
			while(tex1Dfetch(L_tex, j) != tex1Dfetch(L_tex, j+1)){
				// bi = bi - Aij * xj
				index_row = tex1Dfetch(L_tex, j);
				index_col = tex1Dfetch(L_tex, j+1);
				b_x_s[index_row] -= tex1Dfetch(L_tex, j+2) * b_x_s[index_col];
				j += 3;
			}
			i = j;
		}
		
		// doing backward substitution
		i = 3 * L_h_nz - 3;
		while(i >= 0){
			row_p = tex1Dfetch(L_tex, i);
				
			// xi = bi / Aij
			b_x_s[n+row_p] = b_x_s[row_p] / tex1Dfetch(L_tex, i+2);

			j = i-3;
			if(j<0) break;
			//while(L_d[j]!=L_d[j+1]){
			while(tex1Dfetch(L_tex, j) != tex1Dfetch(L_tex, j+1)){
				// bi = bi - Aij * xj
				index_row = tex1Dfetch(L_tex, j);
				index_col = tex1Dfetch(L_tex, j+1);
				b_x_s[index_col] -= tex1Dfetch(L_tex, j+2) * b_x_s[n+index_row];
				j -= 3;
			}
			i = j;
		}
	}
		
	// after computing, copy back into global memory
	for(i=0; i< iter; i++){
		int base = i * blockDim.x;
		if((base+tid) < 2 * n)
			//b_x_d[base+tid] = tex1Dfetch(L_tex, tid);
			b_x_d[base+tid] = b_x_s[base+tid];
	}
}

void substitute_CK_host(float *L_h, size_t L_h_nz, float *bp, float *xp, size_t n){
	float *L_d ;	
	float *b_x_d ;
	// copy data from L to hipArray L_d
	// bind L_d into texture memory
	size_t count = sizeof(float) * 3 * L_h_nz;
	clog<<"L_h_nz: "<<L_h_nz<<endl;
	
	// allocate hipArray and bind it with texture
	cutilSafeCall(hipMalloc((void**)&L_d, count));
	cutilSafeCall(hipMemcpy(L_d, L_h, count, hipMemcpyHostToDevice));
	
	cutilSafeCall(hipBindTexture(0, L_tex, L_d, channelDesc, count));
		
	// malloc b and x into a 1d array, copy from host to device
	count = sizeof(float)* 2 * n;
	cutilSafeCall(hipMalloc((void**)&b_x_d, count));
	size_t index = 0;
	cutilSafeCall(hipMemcpy(b_x_d, bp, sizeof(float)*n, hipMemcpyHostToDevice));
	index += n;
	cutilSafeCall(hipMemcpy(&b_x_d[index], xp, sizeof(float)*n, hipMemcpyHostToDevice));
	
	//substitute_setup(L_h, L_h_nz, L_d, b, x, b_x_d);
	dim3 dimGrid(1, 1);
	dim3 dimBlock(256, 1, 1);
	int sharedMemSize =count;
	clog<<"shared mem size: "<<sharedMemSize<<endl;
	// perform for- and back-ward substitution for each block
	// solution will be written from shared memory into global memory
	substitute_CK_kernel<<<dimGrid, dimBlock, sharedMemSize>>>(L_d, L_h_nz, b_x_d, n);
	cutilCheckMsg("Kernel execution failed.");

	// copy solution back from GPU into CPU
	// where CPU will perform the find_diff and updaterhs()
	//substitute_copy_back(x, b_x_d, b->nrow);
	//substitute_CK_free(L_d, b_x_d);
	//cutilSafeCall(hipMemcpy(bp, b_x_d, sizeof(float)*n, hipMemcpyDeviceToHost));
	cutilSafeCall(hipMemcpy(xp, &b_x_d[n], sizeof(float)*n, hipMemcpyDeviceToHost));
	cutilSafeCall(hipUnbindTexture(L_tex));
	cutilSafeCall(hipFree(L_d));
	cutilSafeCall(hipFree(b_x_d));
}


// copy data from host to device side
// 1. load sparse matrix L from host into global memory, including 4
//    sub-arrays: row index, column index, nz in each column, 
//    and value
// 2. combine L into texture memory
// 3. load dense array b and x from host into global 1d array
void substitute_setup(trip_L *L_h, size_t L_h_nz, float *L_d, cholmod_dense *b, cholmod_dense *&x, double *b_x_d){
	// count is the total bytes of all needed data in L
	// including in order of nz, col, row, and x
	size_t count_unit = sizeof(int)* 2;
	count_unit +=sizeof(double);
	
	size_t count = count_unit * L_h_nz;
	
	// allocate hipArray and bind it with texture
	hipMalloc((void**)&L_d, count);
	hipMemcpy(L_d, L_h, count, hipMemcpyHostToDevice);
	
	// bind L_d to texture memory
	//hipBindTexture(0, L_tex, L_d, count);
		
	// malloc b and x into a 1d array, copy from host to device
	count = sizeof(double)* 2 * b->nrow;
	hipMalloc((void**)&b_x_d, count);
	size_t index = 0;
	hipMemcpy(&b_x_d[0], b->x, b->nrow, hipMemcpyHostToDevice);
	index += b->nrow;
	hipMemcpy(&b_x_d[index], x->x, x->nrow, hipMemcpyHostToDevice);
}

/*
void substitute_copy_back(cholmod_dense *x_h, double *b_x_d, size_t index){
	size_t count = index; // both are nzmax
	hipMemcpy(&x_h->x, &b_x_d[count], count, hipMemcpyDeviceToHost);
}

void substitute_CK_free(float *L_d, double *b_x_d){
	hipUnbindTexture(L_tex);
	hipFree(b_x_d);
	hipFree(L_d);
}*/
