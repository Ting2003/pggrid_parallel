#include "triplet.h"
#include "trip_L.h"
#include <stdio.h>
#include <stdlib.h>
#include "cutil_inline.h"
#include "circuit_host.h"
#include "circuit_kernel.h"
#include "block.h"
#include "global.h"
#include <iostream>
#include <fstream>
using namespace std;

texture <float, 1, hipReadModeElementType> L_tex;
hipChannelFormatDesc channelDesc = 
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	
void substitute_CK_host(trip_L *L_h, size_t L_h_nz, cholmod_dense *b, cholmod_dense *&x){
	float *L_d = NULL;
	double *b_x_d = NULL;
	size_t n = b->nrow;
	// copy data from L to hipArray L_d
	// bind L_d into texture memory
	substitute_setup(L_h, L_h_nz, L_d, b, x, b_x_d);
		
	dim3 dimGrid(1, 1);
	dim3 dimBlock(256, 1, 1);
	//dim3 dimBlock(WARPSIZE, 1, 1);
	// perform for- and back-ward substitution for each block
	// solution will be written from shared memory into global memory
	substitute_CK_kernel<<<dimGrid, dimBlock>>>(L_d, L_h_nz, b_x_d, n);

	// copy solution back from GPU into CPU
	// where CPU will perform the find_diff and updaterhs()
	substitute_copy_back(x, b_x_d, b->nrow);
	substitute_CK_free(L_d, b_x_d);
}

// copy data from host to device side
// 1. load sparse matrix L from host into global memory, including 4
//    sub-arrays: row index, column index, nz in each column, 
//    and value
// 2. combine L into texture memory
// 3. load dense array b and x from host into global 1d array
void substitute_setup(trip_L *L_h, size_t L_h_nz, float *L_d, cholmod_dense *b, cholmod_dense *&x, double *b_x_d){
	// count is the total bytes of all needed data in L
	// including in order of nz, col, row, and x
	size_t count_unit = sizeof(int)* 2;
	count_unit +=sizeof(double);
	
	size_t count = count_unit * L_h_nz;
	
	// allocate hipArray and bind it with texture
	hipMalloc((void **)&L_d, count);
	hipMemcpy(L_d, L_h, count, hipMemcpyHostToDevice);
	
	// bind L_d to texture memory
	hipBindTexture(0, L_tex, L_d, count);
		
	// malloc b and x into a 1d array, copy from host to device
	count = sizeof(double)* 2 * b->nrow;
	hipMalloc((void**)&b_x_d, count);
	size_t index = 0;
	hipMemcpy(&b_x_d[0], b->x, b->nrow, hipMemcpyHostToDevice);
	index += b->nrow;
	hipMemcpy(&b_x_d[index], x->x, x->nrow, hipMemcpyHostToDevice);
}

void substitute_copy_back(cholmod_dense *x_h, double *b_x_d, size_t index){
	size_t count = index; // both are nzmax
	hipMemcpy(&x_h->x, b_x_d, count, hipMemcpyDeviceToHost);
}

void substitute_CK_free(float *L_d, double *b_x_d){
	hipFree(b_x_d);
	hipFree(L_d);
}
